#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include "make_csr.hpp"
#define DEBUG false
#define B_SIZE 1024
#define directed 1
#define weighted 0
#define inf 10000000

struct atomRange {
    long int start, end;
};

struct NonWeightCSR convertToCSR(string fileName) {
    ifstream fin(fileName);
    string line;
    while (getline(fin, line))
    {
        if (line[0] == '%')
        {
            continue;
        }
        else
        {
            break;
        }
    }

    istringstream header(line);
    int num_vertices, num_edges, x;
    header >> num_vertices >> x >> num_edges;
    num_vertices += 1;

    int size;
    if (directed)
        size = num_edges;

    struct NonWeightCSR csr = CSRNonWeighted(num_vertices, num_edges, directed, fin);

    return csr;
}

__global__ void init_dist(int *dist, int vertices) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < vertices) {
        if (id == 0) {
            dist[id] = 0;
        }
        else {
            dist[id] = inf;
        }
    }
}

__global__ void print_dist(int *dist, int num_vertices) {
    for (int i = 0; i < num_vertices; i++) {
        printf("node i = %d, dist = %d\n", i, dist[i]);
    }
}

__device__ struct atomRange getAtomRange(unsigned t_id, long int totalWork, long int totalThreads) {
    long int workToEachThread;
    workToEachThread = totalWork / totalThreads;

    struct atomRange range;
    range.start = t_id * workToEachThread;
    if (t_id == totalThreads - 1) {
        range.end = totalWork;
    }
    else {
        range.end = range.start + workToEachThread;
    }

    return range;
}

__device__ int binarySearch(long int searchItem, long int num_vertices, int *rowOffset) {
    long int start = 0, end = num_vertices - 1, index = end, mid;
    while (start <= end) {
        mid = (start + end) / 2;
        if (rowOffset[mid] > searchItem) {
            end = mid - 1;
        } 
        else {
            index = mid;
            start = mid + 1;
        }
    }

    return index;
}

__global__ void BFS(int *dist, int *src, int *dest, int num_vertices, int num_edges, int *changed) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_vertices) {
        struct atomRange range = getAtomRange(id, num_edges, num_vertices);
        long int u = binarySearch(range.start, num_vertices, src); // get tile

        for (int i = range.start; i < range.end; i++) {
            int v = dest[i];

            // Check if assigned atom goes out of row offset range, if so.. then update the tile
            if (i >= src[u + 1]) {
                u = binarySearch(i, num_vertices, src);
            }

            if(dist[v] > dist[u] + 1){
                atomicMin(&dist[v], dist[u] + 1);
                changed[0] = 1;
            }
        }
        
        // for (int i = src[u]; i < src[u + 1]; i++) {
        //     int v = dest[i];
        //     if(dist[v] > dist[u] + 1){
        //         atomicMin(&dist[v], dist[u] + 1);
        //         changed[0] = 1;
        //     }
        // }
    }
}

int main() {
    // if (argc != 2)
    // {
    //     printf("Usage: %s <input_file>\n", argv[0]);
    //     return 1;
    // }

    // string fileName = argv[1];
    string fileName = "file.txt";
    
    struct NonWeightCSR csr = convertToCSR(fileName);
    int size = csr.num_edges;

    int *dev_row_ptr, *dev_col_ind;
    hipMalloc(&dev_row_ptr, sizeof(int) * (csr.num_vertices + 1));
    hipMalloc(&dev_col_ind, sizeof(int) * size);
    hipMemcpy(dev_row_ptr, csr.row_ptr, sizeof(int) * (csr.num_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ind, csr.col_ind, sizeof(int) * size, hipMemcpyHostToDevice);

    int *dist;
    hipMalloc(&dist, sizeof(int) * csr.num_vertices);

    unsigned nBlocks_for_vertices = ceil((float)csr.num_vertices / B_SIZE);
    init_dist<<<nBlocks_for_vertices, B_SIZE>>>(dist, csr.num_vertices);
    hipDeviceSynchronize();

    int *changed;
    hipMalloc(&changed, sizeof(int));
    hipMallocManaged(&changed, sizeof(int));

    while(true) {
        changed[0] = 0;
        unsigned nBlocks_for_vertices = ceil((float)csr.num_vertices / B_SIZE);

        BFS<<<nBlocks_for_vertices, B_SIZE>>>(dist, dev_row_ptr, dev_col_ind, csr.num_vertices, csr.num_edges, changed);
        hipDeviceSynchronize();

        if (changed[0] == 0) break;
    }

    print_dist<<<1, 1>>>(dist, csr.num_vertices);
    hipDeviceSynchronize();

    return 0;
}