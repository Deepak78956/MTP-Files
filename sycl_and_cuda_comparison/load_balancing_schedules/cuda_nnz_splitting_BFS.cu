#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include "make_csr.hpp"
#define DEBUG false
#define B_SIZE 1024
#define directed 1
#define weighted 0
#define inf 10000000

struct atomRange {
    long int start, end;
};

struct NonWeightCSR convertToCSR(string fileName, bool keywordFound) {
    ifstream fin(fileName);
    string line;
    while (getline(fin, line))
    {
        if (line[0] == '%')
        {
            continue;
        }
        else
        {
            break;
        }
    }

    istringstream header(line);
    int num_vertices, num_edges, x;
    header >> num_vertices >> x >> num_edges;

    int size;
    if (directed)
        size = num_edges;

    struct NonWeightCSR csr = CSRNonWeighted(num_vertices, num_edges, directed, fin, keywordFound);

    return csr;
}

__global__ void init_dist(int *dist, int vertices, int s) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < vertices) {
        if (id == s) {
            dist[id] = 0;
        }
        else {
            dist[id] = inf;
        }
    }
}

__global__ void print_dist(int *dist, int num_vertices) {
    for (int i = 0; i < num_vertices; i++) {
        printf("node i = %d, dist = %d\n", i, dist[i]);
    }
}

__device__ struct atomRange getAtomRange(unsigned t_id, long int totalWork, long int totalThreads, int ttl) {
    long int workToEachThread;
    workToEachThread = totalWork / ttl;

    struct atomRange range;
    range.start = t_id * workToEachThread;
    if (t_id == ttl - 1) {
        range.end = totalWork;
    }
    else {
        range.end = range.start + workToEachThread;
    }

    if (DEBUG) printf("Inside atom range, worktoeachth = %d, id = %d, range = %d %d\n", workToEachThread, t_id, range.start, range.end);

    return range;
}

__device__ int binarySearch(long int searchItem, long int num_vertices, int *rowOffset) {
    long int start = 0, end = num_vertices - 1, index = end, mid;
    while (start <= end) {
        mid = (start + end) / 2;
        if (rowOffset[mid] > searchItem) {
            end = mid - 1;
        } 
        else {
            index = mid;
            start = mid + 1;
        }
    }

    return index;
}

__device__ int updateTileIfReq(int i, int prevTile, int num_vertices, int *src) {
    if (i >= src[prevTile + 1]) {
        prevTile = binarySearch(i, num_vertices, src);
    }

    return prevTile;
}

__global__ void BFS(int *dist, int *src, int *dest, int num_vertices, int num_edges, int *changed, int TTL) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < TTL) {
        struct atomRange range = getAtomRange(id, num_edges, num_vertices, TTL);
        long int u = binarySearch(range.start, num_vertices, src); // get tile

        if (DEBUG) printf("Inside BFS, t_id: %d, index = %d, range = %d %d\n", id, u, range.start, range.end);

        for (int i = range.start; i < range.end; i++) {
            int v = dest[i];

            // Check if assigned atom goes out of row offset range, if so.. then update the tile
            u = updateTileIfReq(i, u, num_vertices, src);

            if(dist[v] > dist[u] + 1){
                atomicMin(&dist[v], dist[u] + 1);
                changed[0] = 1;
            }
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2)
    {
        printf("Usage: %s <input_file>\n", argv[0]);
        return 1;
    }

    string fileName = argv[1];
    // string fileName = "file.txt";

    vector<string> keywords = {"kron", "file"};

    bool keywordFound = false;

    for (const string& keyword : keywords) {
        // Check if the keyword is present in the filename
        if (fileName.find(keyword) != string::npos) {
            // Set the flag to true indicating the keyword is found
            keywordFound = true;
            break;
        }
    }
    
    struct NonWeightCSR csr = convertToCSR(fileName, keywordFound);
    int size = csr.edges;

    int *dev_row_ptr, *dev_col_ind;
    hipMalloc(&dev_row_ptr, sizeof(int) * (csr.vertices + 1));
    hipMalloc(&dev_col_ind, sizeof(int) * size);
    hipMemcpy(dev_row_ptr, csr.row_ptr, sizeof(int) * (csr.vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ind, csr.col_ind, sizeof(int) * size, hipMemcpyHostToDevice);

    int *dist;
    hipMalloc(&dist, sizeof(int) * csr.vertices);

    unsigned nBlocks_for_vertices = ceil((float)csr.vertices / B_SIZE);

    int source = csr.vertices / 2;
    init_dist<<<nBlocks_for_vertices, B_SIZE>>>(dist, csr.vertices, source);
    hipDeviceSynchronize();

    int *changed;
    hipMalloc(&changed, sizeof(int));
    hipMallocManaged(&changed, sizeof(int));

    int TTL = min(csr.vertices, csr.edges);

    clock_t calcTime;
    calcTime = clock();

    while(true) {
        changed[0] = 0;
        unsigned nBlocks_for_vertices = ceil((float)csr.vertices / B_SIZE);

        BFS<<<nBlocks_for_vertices, B_SIZE>>>(dist, dev_row_ptr, dev_col_ind, csr.vertices, csr.edges, changed, TTL);
        hipDeviceSynchronize();

        if (changed[0] == 0) break;
    }

    calcTime = clock() - calcTime;

    double t_time = ((double)calcTime) / CLOCKS_PER_SEC * 1000;

    // print_dist<<<1, 1>>>(dist, csr.vertices);
    // hipDeviceSynchronize();

    // check answer
    int *check_dist;
    check_dist = (int *)malloc(sizeof(int) * csr.vertices);
    for (int i = 0; i < csr.vertices; i++) {
        check_dist[i] = inf;
    }
    check_dist[source] = 0;

    priority_queue<pair<int, int>, vector<pair<int, int>>, greater<pair<int, int>>> pq;
    pq.push({0, source});

    while(!pq.empty()) {
        int u = pq.top().second;
        pq.pop();

        for (int i = csr.row_ptr[u]; i < csr.row_ptr[u + 1]; ++i) {
            int v = csr.col_ind[i];
            int w = 1;

            if (check_dist[u] + w < check_dist[v]) {
                check_dist[v] = check_dist[u] + w;
                pq.push({check_dist[v], v});
            }
        }
    }

    // for (int i = 0; i < csr.vertices; ++i) {
    //     if (check_dist[i] == inf)
    //         cout << "Vertex " << i << ": INF\n";
    //     else
    //         cout << "Vertex " << i << ": " << check_dist[i] << "\n";
    // }

    int *deviceCopiedDist;
    deviceCopiedDist = (int *)malloc(sizeof(int) * csr.vertices);

    hipMemcpy(deviceCopiedDist, dist, sizeof(int) * csr.vertices, hipMemcpyDeviceToHost);

    bool flag = false;
    for (int i = 0; i < csr.vertices; ++i) {
        if (check_dist[i] != deviceCopiedDist[i]) {
            printf("Wrong ans, Expected = %d, Actual = %d on vertex: %d\n", check_dist[i], deviceCopiedDist[i], i);
            flag = true;
            break;
        }
    }
    if (!flag) cout << "Correct ans, Time taken = " << t_time << endl;
    cout << endl;

    return 0;
}