#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include "make_csr.hpp"
#define DEBUG false
#define B_SIZE 1024
#define directed 1
#define weighted 0
#define inf 10000000

struct atomRange {
    long int start, end;
};

struct NonWeightCSR convertToCSR(string fileName) {
    ifstream fin(fileName);
    string line;
    while (getline(fin, line))
    {
        if (line[0] == '%')
        {
            continue;
        }
        else
        {
            break;
        }
    }

    istringstream header(line);
    int num_vertices, num_edges, x;
    header >> num_vertices >> x >> num_edges;

    int size;
    if (directed)
        size = num_edges;

    struct NonWeightCSR csr = CSRNonWeighted(num_vertices, num_edges, directed, fin);

    return csr;
}

__global__ void init_dist(int *dist, int vertices, int s) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < vertices) {
        if (id == s) {
            dist[id] = 0;
        }
        else {
            dist[id] = inf;
        }
    }
}

__global__ void print_dist(int *dist, int num_vertices) {
    for (int i = 0; i < num_vertices; i++) {
        printf("node i = %d, dist = %d\n", i, dist[i]);
    }
}

__device__ struct atomRange getAtomRange(unsigned t_id, long int totalWork, long int totalThreads, int ttl) {
    long int workToEachThread;
    workToEachThread = totalWork / ttl;

    struct atomRange range;
    range.start = t_id * workToEachThread;
    if (t_id == ttl - 1) {
        range.end = totalWork;
    }
    else {
        range.end = range.start + workToEachThread;
    }

    if (DEBUG) printf("Inside atom range, worktoeachth = %d, id = %d, range = %d %d\n", workToEachThread, t_id, range.start, range.end);

    return range;
}

__device__ int binarySearch(long int searchItem, long int num_vertices, int *rowOffset) {
    long int start = 0, end = num_vertices - 1, index = end, mid;
    while (start <= end) {
        mid = (start + end) / 2;
        if (rowOffset[mid] > searchItem) {
            end = mid - 1;
        } 
        else {
            index = mid;
            start = mid + 1;
        }
    }

    return index;
}

__global__ int updateTileIfReq(int i, int prevTile, int num_vertices, int *src) {
    if (i >= src[prevTile + 1]) {
        prevTile = binarySearch(i, num_vertices, src);
    }

    return prevTile;
}

__global__ void BFS(int *dist, int *src, int *dest, int num_vertices, int num_edges, int *changed, int TTL) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < TTL) {
        struct atomRange range = getAtomRange(id, num_edges, num_vertices, TTL);
        long int u = binarySearch(range.start, num_vertices, src); // get tile

        if (DEBUG) printf("Inside BFS, t_id: %d, index = %d, range = %d %d\n", id, u, range.start, range.end);

        for (int i = range.start; i < range.end; i++) {
            int v = dest[i];

            // Check if assigned atom goes out of row offset range, if so.. then update the tile
            u = updateTileIfReq(i, u, num_vertices, src);

            if(dist[v] > dist[u] + 1){
                atomicMin(&dist[v], dist[u] + 1);
                changed[0] = 1;
            }
        }
    }
}

int main() {
    // if (argc != 2)
    // {
    //     printf("Usage: %s <input_file>\n", argv[0]);
    //     return 1;
    // }

    // string fileName = argv[1];
    string fileName = "file.txt";
    
    struct NonWeightCSR csr = convertToCSR(fileName);
    int size = csr.edges;

    int *dev_row_ptr, *dev_col_ind;
    hipMalloc(&dev_row_ptr, sizeof(int) * (csr.vertices + 1));
    hipMalloc(&dev_col_ind, sizeof(int) * size);
    hipMemcpy(dev_row_ptr, csr.row_ptr, sizeof(int) * (csr.vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ind, csr.col_ind, sizeof(int) * size, hipMemcpyHostToDevice);

    int *dist;
    hipMalloc(&dist, sizeof(int) * csr.vertices);

    unsigned nBlocks_for_vertices = ceil((float)csr.vertices / B_SIZE);

    int source = csr.vertices / 2;
    init_dist<<<nBlocks_for_vertices, B_SIZE>>>(dist, csr.vertices, source);
    hipDeviceSynchronize();

    int *changed;
    hipMalloc(&changed, sizeof(int));
    hipMallocManaged(&changed, sizeof(int));

    int TTL = min(csr.vertices, csr.edges);

    while(true) {
        changed[0] = 0;
        unsigned nBlocks_for_vertices = ceil((float)csr.vertices / B_SIZE);

        BFS<<<nBlocks_for_vertices, B_SIZE>>>(dist, dev_row_ptr, dev_col_ind, csr.vertices, csr.edges, changed, TTL);
        hipDeviceSynchronize();

        if (changed[0] == 0) break;
    }

    print_dist<<<1, 1>>>(dist, csr.vertices);
    hipDeviceSynchronize();

    return 0;
}