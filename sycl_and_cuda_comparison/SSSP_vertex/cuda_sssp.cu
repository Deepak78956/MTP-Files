#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include "make_csr.hpp"
#define DEBUG false
#define B_SIZE 1024
#define directed 1
#define weighted 1
#define inf 10000000

__global__ void init_dist(int *dist, int vertices) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < vertices) {
        if (id == 0) {
            dist[id] = 0;
        }
        else {
            dist[id] = 1000000;
        }
    }
}

__global__ void print_dist(int *dist, int num_vertices) {
    for (int i = 0; i < num_vertices; i++) {
        printf("node i = %d, dist = %d\n", i, dist[i]);
    }
}

__global__ void sssp(int *dist, int *src, int *dest, int *weights, int num_vertices, int *changed) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_vertices) {
        int u = id;
        
        for (int i = src[u]; i < src[u + 1]; i++) {
            int v = dest[i];
            int w = weights[i];
            if(dist[v] > dist[u] + w){
                atomicMin(&dist[v], dist[u] + w);
                changed[0] = 1;
            }
        }
    }
}

int main(int argc, char *argv[]) 
{
    // if (argc != 2)
    // {
    //     printf("Usage: %s <input_file>\n", argv[0]);
    //     return 1;
    // }

    // string fileName = argv[1];
    string fileName = "file.txt";
    ifstream fin(fileName);
    string line;
    while (getline(fin, line))
    {
        if (line[0] == '%')
        {
            continue;
        }
        else
        {
            break;
        }
    }

    istringstream header(line);
    int num_vertices, num_edges, x;
    header >> num_vertices >> x >> num_edges;
    // num_vertices += 1;

    int size;
    if (directed)
        size = num_edges;
    
    int *src, *dest, *weights;
    int *dev_src, *dev_dest, *dev_weights;
    src = (int *)malloc(sizeof(int) * (num_vertices + 1));
    dest = (int *)malloc(sizeof(int) * num_edges);
    weights = (int *)malloc(sizeof(int) * num_edges);

    struct WeightCSR csr;
    csr = CSRWeighted(num_vertices, num_edges, directed, fin);

    hipMalloc(&dev_src, sizeof(int) * (num_vertices + 1));
    hipMalloc(&dev_dest, sizeof(int) * num_edges);
    hipMalloc(&dev_weights, sizeof(int) * num_edges);

    for (int i = 0; i < size; i++) {
        dest[i] = csr.col_ind[i];
        weights[i] = csr.weights[i];
    }

    for (int i = 0; i < num_vertices + 1; i++) {
        src[i] = csr.row_ptr[i];
    }

    hipMemcpy(dev_src, src, sizeof(int) * (num_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_dest, dest, sizeof(int) * num_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_weights, weights, sizeof(int) * num_edges, hipMemcpyHostToDevice);

    int *dist;
    hipMalloc(&dist, sizeof(int) * num_vertices);

    unsigned nBlocks_for_vertices = ceil((float)num_vertices / B_SIZE);
    init_dist<<<nBlocks_for_vertices, B_SIZE>>>(dist, num_vertices);
    hipDeviceSynchronize();

    int *changed;
    hipMalloc(&changed, sizeof(int));
    hipMallocManaged(&changed, sizeof(int));
    
    while (true) {
        changed[0] = 0;
        unsigned nBlocks_for_vertices = ceil((float)num_vertices / B_SIZE);
        sssp<<<nBlocks_for_vertices, B_SIZE>>>(dist, dev_src, dev_dest, dev_weights, num_vertices, changed);
        hipDeviceSynchronize();

        if (changed[0] == 0) break;
    }

    // printf("here\n");
    print_dist<<<1, 1>>>(dist, num_vertices);
    hipDeviceSynchronize();

    return 0;
}