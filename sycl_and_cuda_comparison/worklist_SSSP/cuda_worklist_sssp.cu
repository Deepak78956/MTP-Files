#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#include "make_csr.hpp"
#define DEBUG false
#define B_SIZE 1024
#define directed 1
#define weighted 1
#define inf 10000000

int main() {
    // if (argc != 2)
    // {
    //     printf("Usage: %s <input_file>\n", argv[0]);
    //     return 1;
    // }

    // string fileName = argv[1];
    string fileName = "file.txt";
    ifstream fin(fileName);
    string line;
    while (getline(fin, line))
    {
        if (line[0] == '%')
        {
            continue;
        }
        else
        {
            break;
        }
    }

    istringstream header(line);
    int num_vertices, num_edges, x;
    header >> num_vertices >> x >> num_edges;
    // num_vertices += 1;

    int size;
    if (directed)
        size = num_edges;
    
    int *src, *dest, *weights;
    int *dev_src, *dev_dest, *dev_weights;
    src = (int *)malloc(sizeof(int) * (num_vertices + 1));
    dest = (int *)malloc(sizeof(int) * num_edges);
    weights = (int *)malloc(sizeof(int) * num_edges);

    struct WeightCSR csr;
    csr = CSRWeighted(num_vertices, num_edges, directed, fin);

    hipMalloc(&dev_src, sizeof(int) * (num_vertices + 1));
    hipMalloc(&dev_dest, sizeof(int) * num_edges);
    hipMalloc(&dev_weights, sizeof(int) * num_edges);

    for (int i = 0; i < size; i++) {
        dest[i] = csr.col_ind[i];
        weights[i] = csr.weights[i];
    }

    for (int i = 0; i < num_vertices + 1; i++) {
        src[i] = csr.row_ptr[i];
    }

    hipMemcpy(dev_src, src, sizeof(int) * (num_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_dest, dest, sizeof(int) * num_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_weights, weights, sizeof(int) * num_edges, hipMemcpyHostToDevice);

    int *dist;
    hipMalloc(&dist, sizeof(int) * num_vertices);

    unsigned nBlocks_for_vertices = ceil((float)num_vertices / B_SIZE);
    init_dist<<<nBlocks_for_vertices, B_SIZE>>>(dist, num_vertices);
    hipDeviceSynchronize();

    int *changed;
    hipMalloc(&changed, sizeof(int));
    hipMallocManaged(&changed, sizeof(int));

    return 0;
}