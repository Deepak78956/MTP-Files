#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include <hip/hip_runtime.h>
#define DEBUG false
#define B_SIZE 1024
#define directed 1
#define weighted 1
#define inf 10000000

__device__ __managed__ bool changed;

__global__ void init_dist(int *dist, int vertices) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < vertices) {
        if (id == 0) {
            dist[id] = 0;
        }
        else {
            dist[id] = inf;
        }
    }
}

__global__ void sssp(int *dist, int *src, int *dest, int *weights, int num_edges) {
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_edges) {
        int u = src[id];
        int v = dest[id];
        int w = weights[id];
        int newVal = 0;
        atomicAdd(&newVal, dist[u]);
        atomicAdd(&newVal, w);

        if (dist[v] > newVal) {
            dist[v] = newVal;
            changed = true;
        }
    }
}

print_dist(dist, num_vertices);

int main(int argc, char *argv[]) 
{
    if (argc != 2)
    {
        printf("Usage: %s <input_file>\n", argv[0]);
        return 1;
    }

    string fileName = argv[1];
    ifstream fin(fileName);
    string line;
    while (getline(fin, line))
    {
        if (line[0] == '%')
        {
            continue;
        }
        else
        {
            break;
        }
    }

    istringstream header(line);
    int num_vertices, num_edges, x;
    header >> num_vertices >> x >> num_edges;
    // num_vertices += 1;

    int size;
    if (directed)
        size = num_edges;
    
    int *src, *dest, *weights;
    int *dev_src, *dev_dest, *dev_weights;
    src = (int *)malloc(sizeof(int) * num_edges);
    dest = (int *)malloc(sizeof(int) * num_edges);
    weights = (int *)malloc(sizeof(int) * num_edges);
    hipMalloc(&dev_src, sizeof(int) * num_edges);
    hipMalloc(&dev_dest, sizeof(int) * num_edges);
    hipMalloc(&dev_weights, sizeof(int) * num_edges);

    for (int i = 0; i < num_edges; i++) {
        int u, v, w;
        fin >> u >> v >> w;
        src[i] = u - 1;
        dest[i] = v - 1;
        weights[i] = w;
    }

    hipMemcpy(dev_src, src, sizeof(int) * num_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_dest, dest, sizeof(int) * num_edges, hipMemcpyHostToDevice);
    hipMemcpy(dev_weights, weights, sizeof(int) * num_edges, hipMemcpyHostToDevice);

    changed = false;

    int *dist;
    hipMalloc(&dist, sizeof(int) * num_vertices);

    unsigned nBlocks_for_vertices = ceil((float)num_vertices / B_SIZE);
    init_dist<<<nBlocks_for_vertices, B_SIZE>>>(dist, num_vertices);
    hipDeviceSynchronize();

    while (true) {
        unsigned nBlocks_for_edges = ceil((float)num_edges / B_SIZE);
        sssp<<<nBlocks_for_edges, B_SIZE>>>(dist, dev_src, dev_dest, dev_weights, num_edges);
        hipDeviceSynchronize();

        if (changed == false) break;
    }

    print_dist<<<1, 1>>>(dist, num_vertices);
    hipDeviceSynchronize();

    return 0;
}