#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <bits/stdc++.h>
#include <numeric>
#include "make_csr.hpp"
#include <hip/hip_runtime.h>
#define DEBUG false
#define B_SIZE 1024

struct Node
{
    int data;
    struct Node *next;
};

struct Graph
{
    int numVertices;
    struct Node **adjLists;
};

__global__ void initGraph(struct Graph *graph, int vertices, struct Node **adjLists)
{
    graph->numVertices = vertices;
    graph->adjLists = adjLists;

    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;

    if (DEBUG)
    {
        printf("In init graph func\n");
        printf("vertices got %d\n", vertices);
    }

    if (id < vertices)
    {
        graph->adjLists[id] = NULL;
    }
    // for (int i = 0; i < vertices; i++)
    // {
    //     graph->adjLists[i] = NULL;
    // }
    if (DEBUG)
    {
        printf("id = %d and its val %d\n", id, graph->adjLists[id]);
    }
}

__global__ void initEdgeList(struct Node **edgeList, int *dev_col_ind, int size)
{
    if (DEBUG)
    {
        printf("In init edgelist func\n");
        printf("size got %d\n", size);
    }
    // for (int i = 0; i < size; i++)
    // {
    //     if (DEBUG)
    //     {
    //         printf("loop in initEdge %d\n", i);
    //     }

    //     edgeList[i]->data = dev_col_ind[i];

    //     if (DEBUG)
    //     {
    //         printf("data of dev_col_ind i got %d\n", dev_col_ind[i]);
    //         printf("data of Edgelist i got %d\n", edgeList[i]->data);
    //     }
    // }
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        edgeList[id]->data = dev_col_ind[id];
    }
}

__global__ void makeD_LL(struct Node **edgeList, int *dev_row_ptr, struct Graph *graph, int size)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < graph->numVertices)
    {
        int start = dev_row_ptr[id];
        int end = dev_row_ptr[id + 1];
        // printf("For vertex number %d Edges are: ", id);
        for (int v = start; v < end; v++)
        {
            edgeList[v]->next = graph->adjLists[id];
            graph->adjLists[id] = edgeList[v];
            // printf(" %d ", graph->adjLists[id]);
        }
        // printf("\n");
    }
    // for (int u = 0; u < graph->numVertices; u++)
    // {
    //     int start = dev_row_ptr[u];
    //     int end = dev_row_ptr[u + 1];
    //     for (int v = start; v < end; v++)
    //     {
    //         edgeList[v]->next = graph->adjLists[u];
    //         graph->adjLists[u] = edgeList[v];
    //     }
    // }
}

__global__ void printD_LL(struct Graph *graph)
{
    int vertices = graph->numVertices;
    for (int u = 0; u < vertices; u++)
    {
        struct Node *temp = graph->adjLists[u];
        printf("For vertex %d its neighbors are: ", u);
        while (temp)
        {
            printf("%d ", temp->data);
            temp = temp->next;
        }
        printf("\n");
    }
}

__global__ void allocate(struct Node *node, struct Node **edgeList, int i)
{
    edgeList[i] = node;
}

int main()
{
    ifstream fin("file.txt");
    int num_vertices, num_edges, directed, weighted;
    fin >> num_vertices >> num_edges >> directed >> weighted;

    int size;
    if (!directed)
        size = 2 * num_edges;
    else
    {
        cout << "Un-Directed graph is required" << endl;
        exit(0);
    }
    if (weighted)
    {
        cout << "Non weighted graph is required" << endl;
        exit(0);
    }
    struct NonWeightCSR csr = CSRNonWeighted(num_vertices, num_edges, directed, fin);
    if (DEBUG == true)
    {
        for (int i = 0; i < num_vertices + 1; i++)
        {
            cout << csr.row_ptr[i] << " ";
        }
        cout << endl;
        for (int i = 0; i < size; i++)
        {
            cout << csr.col_ind[i] << " ";
        }
        cout << endl;
    }

    int *row_ptr, *col_index;
    row_ptr = (int *)malloc(sizeof(int) * (num_vertices + 1));
    col_index = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < num_vertices + 1; i++)
    {
        row_ptr[i] = csr.row_ptr[i];
    }

    for (int i = 0; i < size; i++)
    {
        col_index[i] = csr.col_ind[i];
    }

    int *dev_row_ptr, *dev_col_ind;
    hipMalloc(&dev_row_ptr, sizeof(int) * (num_vertices + 1));
    hipMalloc(&dev_col_ind, sizeof(int) * size);
    hipMemcpy(dev_row_ptr, row_ptr, sizeof(int) * (num_vertices + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_ind, col_index, sizeof(int) * size, hipMemcpyHostToDevice);

    struct Graph *graph;
    struct Node **adjLists;
    hipMalloc(&graph, sizeof(struct Graph));
    hipMalloc(&adjLists, sizeof(struct Node *) * num_vertices);

    clock_t initClock, initEdgeClock, makeD_LLClock;

    initClock = clock();
    unsigned nBlocks_for_vertices = ceil((float)num_vertices / B_SIZE);
    initGraph<<<nBlocks_for_vertices, B_SIZE>>>(graph, num_vertices, adjLists);
    hipDeviceSynchronize();
    initClock = clock() - initClock;

    struct Node **edgeList;
    hipMalloc(&edgeList, size * sizeof(struct Node *));
    for (int i = 0; i < size; i++)
    {
        struct Node *node;
        hipMalloc(&node, sizeof(struct Node));
        allocate<<<1, 1>>>(node, edgeList, i);
        if (DEBUG)
        {
            printf("Allocated for edge %d\n", i);
        }
    }

    initEdgeClock = clock();
    unsigned nBlocks_for_edges = ceil((float)size / B_SIZE);
    initEdgeList<<<nBlocks_for_edges, B_SIZE>>>(edgeList, dev_col_ind, size);
    hipDeviceSynchronize();
    initEdgeClock = clock() - initEdgeClock;

    makeD_LLClock = clock();
    makeD_LL<<<nBlocks_for_vertices, B_SIZE>>>(edgeList, dev_row_ptr, graph, size);
    hipDeviceSynchronize();
    makeD_LLClock = clock() - makeD_LLClock;

    // printD_LL<<<1, 1>>>(graph);
    // hipDeviceSynchronize();

    cout << endl;
    cout << "Total time taken: " << ((double)(makeD_LLClock + initEdgeClock + initClock)) / CLOCKS_PER_SEC << endl;

    return 0;
}